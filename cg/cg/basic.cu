#include "hip/hip_runtime.h"

#include <stdio.h>
#include<stdlib.h>
#include <string.h>
#include <math.h>

char* concat(char *s1, char *s2);

__global__ void cg_one_global(float* a , int * indeces , float* b , float* x,float * r ,float * r_squared ,float * p_sum ,int size) 
{
	int index = blockDim.x * blockIdx.x + threadIdx.x ;
	float p_temp = 0 ;

	if (index < size)
	{
		float sum = 0 ;
		
		for (int i = 0 ; i<3 ; i++)
		{
			sum += a[3*index  + i] * x[indeces[3*index + i]] ;
		}
		
		r[index] = b[index] - sum ;	
		__syncthreads() ;
		
		for (int i = 0 ; i<3 ; i++)
		{
			p_temp += a[3*index  + i] * r[indeces[3*index + i]] ;

		}
		
		p_sum[index] = p_temp ;
		__syncthreads() ;
		
		r_squared[index] = r[index] * r[index] ;
		p_sum[index] = p_sum[index] * r[index] ;
	}
	
	__syncthreads() ;
	
			
	for (unsigned int s = blockDim.x/2 ; s> 0 ; s >>= 1)
	{	
		if (threadIdx.x < s)
		{
			r_squared[index] = r_squared[index] + r_squared[index +s] ;
			p_sum[index] = p_sum[index] + p_sum[index +s] ;
			__syncthreads() ;
		}
			
	}	

	if (threadIdx.x == 0)
	{
		r_squared[blockIdx.x] = r_squared[index];
		p_sum[blockIdx.x] = p_sum[index];
		__syncthreads() ;
	}
}
__global__ void cg_zero(float* a , int * indeces , float* b , float* x,float * r  ,int size) 
{
	int index = blockDim.x * blockIdx.x + threadIdx.x ;
	int local_index = threadIdx.x ;
	
	if (index < size)
	{
		float sum = 0 ;
		
		for (int i = 0 ; i<3 ; i++)
		{
			sum += a[3*index  + i] * x[indeces[3*index + i]] ;
		}
		
		r[index] = b[index] - sum ;	
	}
	
}

__global__ void cg_one_shared(float* a , int * indeces , float* b , float* x,float * r ,float * r_squared ,float * p_sum ,int size) 
{
	int index = blockDim.x * blockIdx.x + threadIdx.x ;
	int local_index = threadIdx.x ;
	
	__shared__ float shared_r_squared[1024] ;
	__shared__ float shared_p_sum[1024] ;

	shared_r_squared[local_index] = 0 ;
	shared_p_sum[local_index] = 0;
	__syncthreads() ;
	
	if (index < size)
	{
		for (int i = 0 ; i<3 ; i++)
		{
			shared_p_sum[local_index] += a[3*index  + i] * r[indeces[3*index + i]] ;
		}
		__syncthreads() ;

		shared_r_squared[local_index] = r[index] * r[index] ;
		shared_p_sum[local_index] = shared_p_sum[local_index] * r[index] ;
	}
	
	__syncthreads() ;
	for (unsigned int s = blockDim.x/2 ; s> 0 ; s >>= 1)
	{	
		if (threadIdx.x < s)
		{
			shared_r_squared[local_index] = shared_r_squared[local_index] + shared_r_squared[local_index +s] ;
			shared_p_sum[local_index] = shared_p_sum[local_index] + shared_p_sum[local_index +s] ;
			__syncthreads() ;
		}
			
	}	

	if (threadIdx.x == 0)
	{
		r_squared[blockIdx.x] = shared_r_squared[0];
		p_sum[blockIdx.x] = shared_p_sum[0];
		__syncthreads() ;
	}
}

__global__ void cg_two(float * r_squared ,float * p_sum ,int size) 
{
	int index = threadIdx.x ;

	__shared__ float shared_r_squared[1024] ;
	__shared__ float shared_p_sum[1024] ;

	if (index < size)
	{
		shared_r_squared[index] = r_squared[index]  ;
		shared_p_sum[index] = p_sum[index]  ;
	} else
	{
		shared_r_squared[index] = 0 ;
		shared_p_sum[index] = 0 ;
	}
	
	__syncthreads() ;
	
	for (unsigned int s = blockDim.x/2 ; s> 0 ; s >>= 1)
	{	
		if (threadIdx.x < s)
		{
			shared_r_squared[threadIdx.x] = shared_r_squared[threadIdx.x] + shared_r_squared[threadIdx.x +s] ;
			shared_p_sum[threadIdx.x] = shared_p_sum[threadIdx.x] + shared_p_sum[threadIdx.x +s] ;
			__syncthreads() ;
		}	
	}	
	if(threadIdx.x == 0)
	{
		//alpha
		r_squared[blockIdx.x] = shared_r_squared[0]/shared_p_sum[0] ;
		
	}
}

__global__ void cg_three(float * x ,float * r,float * r_squared ,int size) 
{
	int index = blockDim.x * blockIdx.x + threadIdx.x ;
	float alpha = r_squared[0] ;
	x[index] = x[index] + alpha * r[index] ;
}

void cg(const int size , char* file_name)
{
	//initialize our test cases

	float *values = (float *)malloc(3 * size * sizeof(float));
	int *indeces = (int *)malloc(3 * size * sizeof(int));
	float *x = (float *)malloc(size * sizeof(float));
	float *y = (float *)malloc(size * sizeof(float));
	float *output = (float *)malloc(size * sizeof(float));
	float *r_sqaured = (float *)malloc(size * sizeof(float));
	float *p_sum = (float *)malloc(size * sizeof(float));

	char* values_file_name = concat(file_name,"/basic/values.txt") ;
	char* indeces_file_name = concat(file_name,"/basic/indeces.txt");
	char* y_file_name = concat(file_name,"/right_hand_side.txt");
	char* output_file_name = concat(file_name,"/output.txt");

	FILE *values_file = fopen(values_file_name, "r");
	FILE *indeces_file = fopen(indeces_file_name, "r");
	FILE *y_file = fopen(y_file_name, "r");
	FILE *output_file = fopen(output_file_name, "r");

	for (int i = 0 ; i < size ; i++)
	{	
		fscanf(y_file, "%f", &y[i]);
		fscanf(output_file, "%f", &output[i]);
		x[i] = 0 ;
	}

	for (int i = 0 ; i< size; i++)
	{
		r_sqaured[i] = 0 ;
		p_sum[i] = 0 ;
	}

	for (int i = 0 ; i< 3 * size ; i++)
	{
		fscanf(values_file, "%f", &values[i]);
		fscanf(indeces_file, "%d", &indeces[i]);	
	}
	
	float* dev_values = 0;
	int* dev_indeces = 0 ;
	float* dev_y = 0;
	float* dev_x = 0;
	float* dev_r = 0 ;
	float* dev_r_squared = 0 ;
	float* dev_p_sum = 0;

	int number_of_blocks = 10 ;
	int number_of_threads = 42 ;
    hipSetDevice(0);
	
    // Allocate GPU buffers
    hipMalloc((void**)&dev_values, 3 * size * sizeof(float));
	hipMalloc((void**)&dev_indeces, 3 * size * sizeof(int));
    hipMalloc((void**)&dev_y, size * sizeof(float));
    hipMalloc((void**)&dev_x, size * sizeof(float));
	hipMalloc((void**)&dev_r, size * sizeof(float));
	hipMalloc((void**)&dev_r_squared, number_of_blocks * sizeof(float));
	hipMalloc((void**)&dev_p_sum, number_of_blocks * sizeof(float));
   
    // Copy input vectors from host memory to GPU buffers.
	hipMemcpyAsync(dev_values, values, 3 * size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyAsync(dev_indeces, indeces, 3 * size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpyAsync(dev_y, y, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyAsync(dev_x, x, size * sizeof(float), hipMemcpyHostToDevice);
	
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

    // Launch a kernel on the GPU with one thread for each row.
	cg_zero<<<number_of_blocks,number_of_threads>>>(dev_values , dev_indeces , dev_y ,  dev_x, dev_r , size) ;
	cg_one_shared<<<number_of_blocks,number_of_threads>>>(dev_values , dev_indeces , dev_y ,  dev_x, dev_r , dev_r_squared , dev_p_sum , size) ;
	cg_two<<<1,number_of_blocks>>>(dev_r_squared ,dev_p_sum ,number_of_blocks);
	cg_three<<<number_of_blocks,number_of_threads>>>( dev_x ,dev_r,dev_r_squared , size);
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf ("Time for the kernel: %f ms\n", time);

    // Copy output vector from GPU buffer to host memory.
    hipMemcpy(x, dev_x, size * sizeof(float), hipMemcpyDeviceToHost);

	printf("%f\n",x[0]);
	printf("%f\n",x[1]);
	printf("%f\n",x[2]);
	printf("%f\n",x[size -2]);
	printf("%f\n",x[size -1]);
	
	hipFree(dev_values);
	hipFree(dev_indeces) ;
	hipFree(dev_y);
	hipFree(dev_x);
	hipFree(dev_r) ;
	hipFree(dev_r_squared) ;
	hipFree(dev_p_sum) ;
	hipDeviceReset();
	system("pause");
}

char* concat(char *s1, char *s2)
{
    char *result = (char *)malloc(strlen(s1)+strlen(s2)+1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

int main()
{
	cg(420,"C:/Users/youssef/Desktop/numerical-solutions-gpu/cg/cg/test_cases/420");
	return 1 ;
}