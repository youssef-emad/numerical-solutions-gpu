#include "hip/hip_runtime.h"

#include <stdio.h>
#include<stdlib.h>
#include <string.h>
#include <math.h>

char* concat(char *s1, char *s2);

__global__  void cg_global(float* a , int * indeces , float* b , float* x,int size) 
{
	int index = threadIdx.x ;
	__shared__  float r[1000] ;
	__shared__  float r_squared[1000] ;
	__shared__  float p_sum[1000] ;

	if (index < size)
	{
		float sum = 0 ;

		for (int i = 0 ; i<3 ; i++)
		{
			sum += a[i + 3*index] * x[indeces[i + 3*index]] ;
		}
		
		r[index] = b[index] - sum ;	
		p_sum[index] = 0 ;
		__syncthreads();

		for (int i = 0 ; i<3 ; i++)
		{
			p_sum[index] += a[i + 3*index] * r[indeces[i + 3*index]] ;
		}
		
		//calc alpha
		r_squared[index] = r[index] * r[index] ;
		p_sum[index] = p_sum[index] * r[index] ;

		for (unsigned int s = blockDim.x/2 ; s> 0 ; s >>= 1)
		{
			if (index < size/2)
			{
				// summation of r*rT
				r_squared[index] = r_squared[index] + r_squared[index + s] ;
				//summation of r*a*rT
				p_sum[index] = p_sum[index] +  p_sum[index + s] ;
			}
			__syncthreads();
		}

		float alpha = r_squared[0]/p_sum[0] ;
		
		x[index] = x[index] + alpha * r[index] ;

	}

}

__global__ void cg_local(float* a , int * indeces , float* b , float* x,int size) 
{
	int index = threadIdx.x ;
	float local_r ;
	float local_a[3] = {a[3*index],a[3*index + 1],a[3*index +2]} ;
	int local_indeces[3] = {indeces[3*index],indeces[3*index + 1],indeces[3*index +2]} ;

	__shared__ float r[100] ;
	__shared__  float r_squared[100] ;
	__shared__  float p_sum[100] ;

	if (index < size)
	{
		float sum = 0 ;

		for (int i = 0 ; i<3 ; i++)
		{
			sum += local_a[i] * x[local_indeces[i]] ;
		}
		
		local_r = b[index] - sum ;
		r[index]  = local_r;
		p_sum[index] = 0 ;

		__syncthreads();

		for (int i = 0 ; i<3 ; i++)
		{
			p_sum[index] += local_a[i] * r[local_indeces[i]] ;
		}
		
		//calc alpha
		r_squared[index] = local_r * local_r ;
		p_sum[index] = p_sum[index] * local_r ;

		for (unsigned int s = blockDim.x/2 ; s> 0 ; s >>= 1)
		{
			if (index < size/2)
			{
				// summation of r*rT
				r_squared[index] = r_squared[2*index] + r_squared[2*index + 1] ;
				//summation of r*a*rT
				p_sum[index] = p_sum[2*index] +  p_sum[2*index + 1] ;
			}
			__syncthreads();
		}
	
		float alpha = r_squared[0]/p_sum[0] ;
		x[index] = x[index] + alpha * local_r ;

	}

}


void cg(const int size , char* file_name)
{
	//initialize our test cases

	float *values = (float *)malloc(3 * size * sizeof(float));
	int *indeces = (int *)malloc(3 * size * sizeof(int));
	float *x = (float *)malloc(size * sizeof(float));
	float *y = (float *)malloc(size * sizeof(float));
	float *output = (float *)malloc(size * sizeof(float));

	char* values_file_name = concat(file_name,"/basic/values.txt") ;
	char* indeces_file_name = concat(file_name,"/basic/indeces.txt");
	char* y_file_name = concat(file_name,"/right_hand_side.txt");
	char* output_file_name = concat(file_name,"/output.txt");

	FILE *values_file = fopen(values_file_name, "r");
	FILE *indeces_file = fopen(indeces_file_name, "r");
	FILE *y_file = fopen(y_file_name, "r");
	FILE *output_file = fopen(output_file_name, "r");

	for (int i = 0 ; i < size ; i++)
	{	
		fscanf(y_file, "%f", &y[i]);
		fscanf(output_file, "%f", &output[i]);
		x[i] = y[i] ;
	}

	for (int i = 0 ; i< 3 * size ; i++)
	{
		fscanf(values_file, "%f", &values[i]);
		fscanf(indeces_file, "%d", &indeces[i]);	
	}
	
	float* dev_values = 0;
	int* dev_indeces = 0 ;
	float* dev_y = 0;
	float* dev_x = 0;

    hipSetDevice(0);
	
    // Allocate GPU buffers
   
    hipMalloc((void**)&dev_values, 3 * size * sizeof(float));
	hipMalloc((void**)&dev_indeces, 3 * size * sizeof(int));
    hipMalloc((void**)&dev_y, size * sizeof(float));
    hipMalloc((void**)&dev_x, size * sizeof(float));
   
    // Copy input vectors from host memory to GPU buffers.
	hipMemcpyAsync(dev_values, values, 3 * size * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyAsync(dev_indeces, indeces, 3 * size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpyAsync(dev_y, y, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyAsync(dev_x, x, size * sizeof(float), hipMemcpyHostToDevice);
	
	hipEvent_t start, stop;
	float time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

    // Launch a kernel on the GPU with one thread for each row.
	cg_global<<<1,size>>>(dev_values,dev_indeces,dev_y,dev_x,size);
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
	hipDeviceSynchronize();
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	printf ("Time for the kernel: %f ms\n", time);

    // Copy output vector from GPU buffer to host memory.
    hipMemcpyAsync(x, dev_x, size * sizeof(float), hipMemcpyDeviceToHost);
	
	printf("%f\n",x[0]);
	printf("%f\n",x[1]);
	printf("%f\n",x[2]);
	hipDeviceReset();
	system("pause");
}

char* concat(char *s1, char *s2)
{
    char *result = (char *)malloc(strlen(s1)+strlen(s2)+1);
    strcpy(result, s1);
    strcat(result, s2);
    return result;
}

int main()
{
	cg(420,"C:/Users/youssef/Desktop/numerical-solutions-gpu/cg/cg/test_cases/420");
	return 1 ;
}